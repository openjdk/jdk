#include "hip/hip_runtime.h"
//   Copyright Naoki Shibata and contributors 2010 - 2021.
// Distributed under the Boost Software License, Version 1.0.
//    (See accompanying file LICENSE.txt or copy at
//          http://www.boost.org/LICENSE_1_0.txt)

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <math.h>
#include <float.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#include "sleefinline_purec_scalar.h"
#include "sleefinline_cuda.h"

#define STDIN_FILENO 0

#define SIMD_SUFFIX _cuda_sleef
#define CONCAT_SIMD_SUFFIX_(keyword, suffix) keyword ## suffix
#define CONCAT_SIMD_SUFFIX(keyword, suffix) CONCAT_SIMD_SUFFIX_(keyword, suffix)

#define vdouble2 CONCAT_SIMD_SUFFIX(vdouble2, SIMD_SUFFIX)
#define vfloat2 CONCAT_SIMD_SUFFIX(vfloat2, SIMD_SUFFIX)

//

static int startsWith(const char *str, const char *prefix) {
  while(*prefix != '\0') if (*str++ != *prefix++) return 0;
  return *prefix == '\0';
}

static double u2d(uint64_t u) {
  union {
    double f;
    uint64_t i;
  } tmp;
  tmp.i = u;
  return tmp.f;
}

static uint64_t d2u(double d) {
  union {
    double f;
    uint64_t i;
  } tmp;
  tmp.f = d;
  return tmp.i;
}

static float u2f(uint32_t u) {
  union {
    float f;
    uint32_t i;
  } tmp;
  tmp.i = u;
  return tmp.f;
}

static uint32_t f2u(float d) {
  union {
    float f;
    uint32_t i;
  } tmp;
  tmp.f = d;
  return tmp.i;
}

//

__global__ void xsin(double *r, double *a0) { *r = Sleef_sind1_u35cuda(*a0); }
__global__ void xcos(double *r, double *a0) { *r = Sleef_cosd1_u35cuda(*a0); }
__global__ void xsincos(vdouble2 *r, double *a0) { *r = Sleef_sincosd1_u35cuda(*a0); }
__global__ void xtan(double *r, double *a0) { *r = Sleef_tand1_u35cuda(*a0); }
__global__ void xasin(double *r, double *a0) { *r = Sleef_asind1_u35cuda(*a0); }
__global__ void xacos(double *r, double *a0) { *r = Sleef_acosd1_u35cuda(*a0); }
__global__ void xatan(double *r, double *a0) { *r = Sleef_atand1_u35cuda(*a0); }
__global__ void xatan2(double *r, double *a0, double *a1) { *r = Sleef_atan2d1_u35cuda(*a0, *a1); }
__global__ void xlog(double *r, double *a0) { *r = Sleef_logd1_u35cuda(*a0); }
__global__ void xcbrt(double *r, double *a0) { *r = Sleef_cbrtd1_u35cuda(*a0); }
__global__ void xsin_u1(double *r, double *a0) { *r = Sleef_sind1_u10cuda(*a0); }
__global__ void xcos_u1(double *r, double *a0) { *r = Sleef_cosd1_u10cuda(*a0); }
__global__ void xsincos_u1(vdouble2 *r, double *a0) { *r = Sleef_sincosd1_u10cuda(*a0); }
__global__ void xtan_u1(double *r, double *a0) { *r = Sleef_tand1_u10cuda(*a0); }
__global__ void xasin_u1(double *r, double *a0) { *r = Sleef_asind1_u10cuda(*a0); }
__global__ void xacos_u1(double *r, double *a0) { *r = Sleef_acosd1_u10cuda(*a0); }
__global__ void xatan_u1(double *r, double *a0) { *r = Sleef_atand1_u10cuda(*a0); }
__global__ void xatan2_u1(double *r, double *a0, double *a1) { *r = Sleef_atan2d1_u10cuda(*a0, *a1); }
__global__ void xlog_u1(double *r, double *a0) { *r = Sleef_logd1_u10cuda(*a0); }
__global__ void xcbrt_u1(double *r, double *a0) { *r = Sleef_cbrtd1_u10cuda(*a0); }
__global__ void xexp(double *r, double *a0) { *r = Sleef_expd1_u10cuda(*a0); }
__global__ void xpow(double *r, double *a0, double *a1) { *r = Sleef_powd1_u10cuda(*a0, *a1); }
__global__ void xsinh(double *r, double *a0) { *r = Sleef_sinhd1_u10cuda(*a0); }
__global__ void xcosh(double *r, double *a0) { *r = Sleef_coshd1_u10cuda(*a0); }
__global__ void xtanh(double *r, double *a0) { *r = Sleef_tanhd1_u10cuda(*a0); }
__global__ void xsinh_u35(double *r, double *a0) { *r = Sleef_sinhd1_u35cuda(*a0); }
__global__ void xcosh_u35(double *r, double *a0) { *r = Sleef_coshd1_u35cuda(*a0); }
__global__ void xtanh_u35(double *r, double *a0) { *r = Sleef_tanhd1_u35cuda(*a0); }
__global__ void xasinh(double *r, double *a0) { *r = Sleef_asinhd1_u10cuda(*a0); }
__global__ void xacosh(double *r, double *a0) { *r = Sleef_acoshd1_u10cuda(*a0); }
__global__ void xatanh(double *r, double *a0) { *r = Sleef_atanhd1_u10cuda(*a0); }
__global__ void xexp2(double *r, double *a0) { *r = Sleef_exp2d1_u10cuda(*a0); }
__global__ void xexp2_u35(double *r, double *a0) { *r = Sleef_exp2d1_u35cuda(*a0); }
__global__ void xexp10(double *r, double *a0) { *r = Sleef_exp10d1_u10cuda(*a0); }
__global__ void xexp10_u35(double *r, double *a0) { *r = Sleef_exp10d1_u35cuda(*a0); }
__global__ void xexpm1(double *r, double *a0) { *r = Sleef_expm1d1_u10cuda(*a0); }
__global__ void xlog10(double *r, double *a0) { *r = Sleef_log10d1_u10cuda(*a0); }
__global__ void xlog2(double *r, double *a0) { *r = Sleef_log2d1_u10cuda(*a0); }
__global__ void xlog2_u35(double *r, double *a0) { *r = Sleef_log2d1_u35cuda(*a0); }
__global__ void xlog1p(double *r, double *a0) { *r = Sleef_log1pd1_u10cuda(*a0); }
__global__ void xsincospi_u05(vdouble2 *r, double *a0) { *r = Sleef_sincospid1_u05cuda(*a0); }
__global__ void xsincospi_u35(vdouble2 *r, double *a0) { *r = Sleef_sincospid1_u35cuda(*a0); }
__global__ void xsinpi_u05(double *r, double *a0) { *r = Sleef_sinpid1_u05cuda(*a0); }
__global__ void xcospi_u05(double *r, double *a0) { *r = Sleef_cospid1_u05cuda(*a0); }
__global__ void xldexp(double *r, double *a0, int *a1) { *r = Sleef_ldexpd1_cuda(*a0, *a1); }
__global__ void xilogb(int *r, double *a0) { *r = Sleef_ilogbd1_cuda(*a0); }
__global__ void xfma(double *r, double *a0, double *a1, double *a2) { *r = Sleef_fmad1_cuda(*a0, *a1, *a2); }
__global__ void xsqrt(double *r, double *a0) { *r = Sleef_sqrtd1_cuda(*a0); }
__global__ void xsqrt_u05(double *r, double *a0) { *r = Sleef_sqrtd1_u05cuda(*a0); }
__global__ void xsqrt_u35(double *r, double *a0) { *r = Sleef_sqrtd1_u35cuda(*a0); }
__global__ void xhypot_u05(double *r, double *a0, double *a1) { *r = Sleef_hypotd1_u05cuda(*a0, *a1); }
__global__ void xhypot_u35(double *r, double *a0, double *a1) { *r = Sleef_hypotd1_u35cuda(*a0, *a1); }
__global__ void xfabs(double *r, double *a0) { *r = Sleef_fabsd1_cuda(*a0); }
__global__ void xcopysign(double *r, double *a0, double *a1) { *r = Sleef_copysignd1_cuda(*a0, *a1); }
__global__ void xfmax(double *r, double *a0, double *a1) { *r = Sleef_fmaxd1_cuda(*a0, *a1); }
__global__ void xfmin(double *r, double *a0, double *a1) { *r = Sleef_fmind1_cuda(*a0, *a1); }
__global__ void xfdim(double *r, double *a0, double *a1) { *r = Sleef_fdimd1_cuda(*a0, *a1); }
__global__ void xtrunc(double *r, double *a0) { *r = Sleef_truncd1_cuda(*a0); }
__global__ void xfloor(double *r, double *a0) { *r = Sleef_floord1_cuda(*a0); }
__global__ void xceil(double *r, double *a0) { *r = Sleef_ceild1_cuda(*a0); }
__global__ void xround(double *r, double *a0) { *r = Sleef_roundd1_cuda(*a0); }
__global__ void xrint(double *r, double *a0) { *r = Sleef_rintd1_cuda(*a0); }
__global__ void xnextafter(double *r, double *a0, double *a1) { *r = Sleef_nextafterd1_cuda(*a0, *a1); }
__global__ void xfrfrexp(double *r, double *a0) { *r = Sleef_frfrexpd1_cuda(*a0); }
__global__ void xexpfrexp(int *r, double *a0) { *r = Sleef_expfrexpd1_cuda(*a0); }
__global__ void xfmod(double *r, double *a0, double *a1) { *r = Sleef_fmodd1_cuda(*a0, *a1); }
__global__ void xremainder(double *r, double *a0, double *a1) { *r = Sleef_remainderd1_cuda(*a0, *a1); }
__global__ void xmodf(vdouble2 *r, double *a0) { *r = Sleef_modfd1_cuda(*a0); }
__global__ void xlgamma_u1(double *r, double *a0) { *r = Sleef_lgammad1_u10cuda(*a0); }
__global__ void xtgamma_u1(double *r, double *a0) { *r = Sleef_tgammad1_u10cuda(*a0); }
__global__ void xerf_u1(double *r, double *a0) { *r = Sleef_erfd1_u10cuda(*a0); }
__global__ void xerfc_u15(double *r, double *a0) { *r = Sleef_erfcd1_u15cuda(*a0); }

__global__ void xsinf(float *r, float *a0) { *r = Sleef_sinf1_u35cuda(*a0); }
__global__ void xcosf(float *r, float *a0) { *r = Sleef_cosf1_u35cuda(*a0); }
__global__ void xsincosf(vfloat2 *r, float *a0) { *r = Sleef_sincosf1_u35cuda(*a0); }
__global__ void xtanf(float *r, float *a0) { *r = Sleef_tanf1_u35cuda(*a0); }
__global__ void xasinf(float *r, float *a0) { *r = Sleef_asinf1_u35cuda(*a0); }
__global__ void xacosf(float *r, float *a0) { *r = Sleef_acosf1_u35cuda(*a0); }
__global__ void xatanf(float *r, float *a0) { *r = Sleef_atanf1_u35cuda(*a0); }
__global__ void xatan2f(float *r, float *a0, float *a1) { *r = Sleef_atan2f1_u35cuda(*a0, *a1); }
__global__ void xlogf(float *r, float *a0) { *r = Sleef_logf1_u35cuda(*a0); }
__global__ void xcbrtf(float *r, float *a0) { *r = Sleef_cbrtf1_u35cuda(*a0); }
__global__ void xsinf_u1(float *r, float *a0) { *r = Sleef_sinf1_u10cuda(*a0); }
__global__ void xcosf_u1(float *r, float *a0) { *r = Sleef_cosf1_u10cuda(*a0); }
__global__ void xsincosf_u1(vfloat2 *r, float *a0) { *r = Sleef_sincosf1_u10cuda(*a0); }
__global__ void xtanf_u1(float *r, float *a0) { *r = Sleef_tanf1_u10cuda(*a0); }
__global__ void xasinf_u1(float *r, float *a0) { *r = Sleef_asinf1_u10cuda(*a0); }
__global__ void xacosf_u1(float *r, float *a0) { *r = Sleef_acosf1_u10cuda(*a0); }
__global__ void xatanf_u1(float *r, float *a0) { *r = Sleef_atanf1_u10cuda(*a0); }
__global__ void xatan2f_u1(float *r, float *a0, float *a1) { *r = Sleef_atan2f1_u10cuda(*a0, *a1); }
__global__ void xlogf_u1(float *r, float *a0) { *r = Sleef_logf1_u10cuda(*a0); }
__global__ void xcbrtf_u1(float *r, float *a0) { *r = Sleef_cbrtf1_u10cuda(*a0); }
__global__ void xexpf(float *r, float *a0) { *r = Sleef_expf1_u10cuda(*a0); }
__global__ void xpowf(float *r, float *a0, float *a1) { *r = Sleef_powf1_u10cuda(*a0, *a1); }
__global__ void xsinhf(float *r, float *a0) { *r = Sleef_sinhf1_u10cuda(*a0); }
__global__ void xcoshf(float *r, float *a0) { *r = Sleef_coshf1_u10cuda(*a0); }
__global__ void xtanhf(float *r, float *a0) { *r = Sleef_tanhf1_u10cuda(*a0); }
__global__ void xsinhf_u35(float *r, float *a0) { *r = Sleef_sinhf1_u35cuda(*a0); }
__global__ void xcoshf_u35(float *r, float *a0) { *r = Sleef_coshf1_u35cuda(*a0); }
__global__ void xtanhf_u35(float *r, float *a0) { *r = Sleef_tanhf1_u35cuda(*a0); }
__global__ void xfastsinf_u3500(float *r, float *a0) { *r = Sleef_fastsinf1_u3500cuda(*a0); }
__global__ void xfastcosf_u3500(float *r, float *a0) { *r = Sleef_fastcosf1_u3500cuda(*a0); }
__global__ void xfastpowf_u3500(float *r, float *a0, float *a1) { *r = Sleef_fastpowf1_u3500cuda(*a0, *a1); }
__global__ void xasinhf(float *r, float *a0) { *r = Sleef_asinhf1_u10cuda(*a0); }
__global__ void xacoshf(float *r, float *a0) { *r = Sleef_acoshf1_u10cuda(*a0); }
__global__ void xatanhf(float *r, float *a0) { *r = Sleef_atanhf1_u10cuda(*a0); }
__global__ void xexp2f(float *r, float *a0) { *r = Sleef_exp2f1_u10cuda(*a0); }
__global__ void xexp2f_u35(float *r, float *a0) { *r = Sleef_exp2f1_u35cuda(*a0); }
__global__ void xexp10f(float *r, float *a0) { *r = Sleef_exp10f1_u10cuda(*a0); }
__global__ void xexp10f_u35(float *r, float *a0) { *r = Sleef_exp10f1_u35cuda(*a0); }
__global__ void xexpm1f(float *r, float *a0) { *r = Sleef_expm1f1_u10cuda(*a0); }
__global__ void xlog10f(float *r, float *a0) { *r = Sleef_log10f1_u10cuda(*a0); }
__global__ void xlog2f(float *r, float *a0) { *r = Sleef_log2f1_u10cuda(*a0); }
__global__ void xlog2f_u35(float *r, float *a0) { *r = Sleef_log2f1_u35cuda(*a0); }
__global__ void xlog1pf(float *r, float *a0) { *r = Sleef_log1pf1_u10cuda(*a0); }
__global__ void xsincospif_u05(vfloat2 *r, float *a0) { *r = Sleef_sincospif1_u05cuda(*a0); }
__global__ void xsincospif_u35(vfloat2 *r, float *a0) { *r = Sleef_sincospif1_u35cuda(*a0); }
__global__ void xsinpif_u05(float *r, float *a0) { *r = Sleef_sinpif1_u05cuda(*a0); }
__global__ void xcospif_u05(float *r, float *a0) { *r = Sleef_cospif1_u05cuda(*a0); }
__global__ void xldexpf(float *r, float *a0, int *a1) { *r = Sleef_ldexpf1_cuda(*a0, *a1); }
__global__ void xilogbf(int *r, float *a0) { *r = Sleef_ilogbf1_cuda(*a0); }
__global__ void xfmaf(float *r, float *a0, float *a1, float *a2) { *r = Sleef_fmaf1_cuda(*a0, *a1, *a2); }
__global__ void xsqrtf(float *r, float *a0) { *r = Sleef_sqrtf1_cuda(*a0); }
__global__ void xsqrtf_u05(float *r, float *a0) { *r = Sleef_sqrtf1_u05cuda(*a0); }
__global__ void xsqrtf_u35(float *r, float *a0) { *r = Sleef_sqrtf1_u35cuda(*a0); }
__global__ void xhypotf_u05(float *r, float *a0, float *a1) { *r = Sleef_hypotf1_u05cuda(*a0, *a1); }
__global__ void xhypotf_u35(float *r, float *a0, float *a1) { *r = Sleef_hypotf1_u35cuda(*a0, *a1); }
__global__ void xfabsf(float *r, float *a0) { *r = Sleef_fabsf1_cuda(*a0); }
__global__ void xcopysignf(float *r, float *a0, float *a1) { *r = Sleef_copysignf1_cuda(*a0, *a1); }
__global__ void xfmaxf(float *r, float *a0, float *a1) { *r = Sleef_fmaxf1_cuda(*a0, *a1); }
__global__ void xfminf(float *r, float *a0, float *a1) { *r = Sleef_fminf1_cuda(*a0, *a1); }
__global__ void xfdimf(float *r, float *a0, float *a1) { *r = Sleef_fdimf1_cuda(*a0, *a1); }
__global__ void xtruncf(float *r, float *a0) { *r = Sleef_truncf1_cuda(*a0); }
__global__ void xfloorf(float *r, float *a0) { *r = Sleef_floorf1_cuda(*a0); }
__global__ void xceilf(float *r, float *a0) { *r = Sleef_ceilf1_cuda(*a0); }
__global__ void xroundf(float *r, float *a0) { *r = Sleef_roundf1_cuda(*a0); }
__global__ void xrintf(float *r, float *a0) { *r = Sleef_rintf1_cuda(*a0); }
__global__ void xnextafterf(float *r, float *a0, float *a1) { *r = Sleef_nextafterf1_cuda(*a0, *a1); }
__global__ void xfrfrexpf(float *r, float *a0) { *r = Sleef_frfrexpf1_cuda(*a0); }
__global__ void xexpfrexpf(float *r, float *a0) { *r = Sleef_expfrexpf1_cuda(*a0); }
__global__ void xfmodf(float *r, float *a0, float *a1) { *r = Sleef_fmodf1_cuda(*a0, *a1); }
__global__ void xremainderf(float *r, float *a0, float *a1) { *r = Sleef_remainderf1_cuda(*a0, *a1); }
__global__ void xmodff(vfloat2 *r, float *a0) { *r = Sleef_modff1_cuda(*a0); }
__global__ void xlgammaf_u1(float *r, float *a0) { *r = Sleef_lgammaf1_u10cuda(*a0); }
__global__ void xtgammaf_u1(float *r, float *a0) { *r = Sleef_tgammaf1_u10cuda(*a0); }
__global__ void xerff_u1(float *r, float *a0) { *r = Sleef_erff1_u10cuda(*a0); }
__global__ void xerfcf_u15(float *r, float *a0) { *r = Sleef_erfcf1_u15cuda(*a0); }

//

#define func_d_d(funcStr, funcName) {                           \
    while (startsWith(buf, funcStr " ")) {                      \
      uint64_t u;                                               \
      sscanf(buf, funcStr " %" PRIx64, &u);                     \
      *a0 = u2d(u);                                             \
      funcName<<<1, 1>>>(r, a0);                                \
      hipDeviceSynchronize();                                  \
      printf("%" PRIx64 "\n", d2u(*r));                         \
      fflush(stdout);                                           \
      if (fgets(buf, BUFSIZE-1, stdin) == NULL) break;          \
    }                                                           \
  }

#define func_d2_d(funcStr, funcName) {                                  \
    while (startsWith(buf, funcStr " ")) {                              \
      uint64_t u;                                                       \
      sscanf(buf, funcStr " %" PRIx64, &u);                             \
      *a0 = u2d(u);                                                     \
      funcName<<<1, 1>>>(r2, a0);                                       \
      hipDeviceSynchronize();                                          \
      printf("%" PRIx64 " %" PRIx64 "\n", d2u(r2->x), d2u(r2->y));      \
      fflush(stdout);                                                   \
      if (fgets(buf, BUFSIZE-1, stdin) == NULL) break;                  \
    }                                                                   \
  }

#define func_d_d_d(funcStr, funcName) {                         \
    while (startsWith(buf, funcStr " ")) {                      \
      uint64_t u, v;                                            \
      sscanf(buf, funcStr " %" PRIx64 " %" PRIx64, &u, &v);     \
      *a0 = u2d(u);                                             \
      *a1 = u2d(v);                                             \
      funcName<<<1, 1>>>(r, a0, a1);                            \
      hipDeviceSynchronize();                                  \
      printf("%" PRIx64 "\n", d2u(*r));                         \
      fflush(stdout);                                           \
      if (fgets(buf, BUFSIZE-1, stdin) == NULL) break;          \
    }                                                           \
  }

#define func_d_d_i(funcStr, funcName) {                                 \
    while (startsWith(buf, funcStr " ")) {                              \
      uint64_t u, v;                                                    \
      sscanf(buf, funcStr " %" PRIx64 " %" PRIx64, &u, &v);             \
      *a0 = u2d(u);                                                     \
      *i0 = (int)u2d(v);                                                \
      funcName<<<1, 1>>>(r, a0, i0);                                    \
      hipDeviceSynchronize();                                          \
      printf("%" PRIx64 "\n", d2u(*r));                                 \
      fflush(stdout);                                                   \
      if (fgets(buf, BUFSIZE-1, stdin) == NULL) break;                  \
    }                                                                   \
  }

#define func_i_d(funcStr, funcName) {                   \
    while (startsWith(buf, funcStr " ")) {              \
      uint64_t u;                                       \
      sscanf(buf, funcStr " %" PRIx64, &u);             \
      *a0 = u2d(u);                                     \
      funcName<<<1, 1>>>(i0, a0);                       \
      hipDeviceSynchronize();                          \
      printf("%d\n", *i0);                              \
      fflush(stdout);                                   \
      if (fgets(buf, BUFSIZE-1, stdin) == NULL) break;  \
    }                                                   \
  }

//

#define func_f_f(funcStr, funcName) {                           \
    while (startsWith(buf, funcStr " ")) {                      \
      uint32_t u;                                               \
      sscanf(buf, funcStr " %x", &u);                           \
      *b0 = u2f(u);                                             \
      funcName<<<1, 1>>>(s, b0);                                \
      hipDeviceSynchronize();                                  \
      printf("%x\n", f2u(*s));                                  \
      fflush(stdout);                                           \
      if (fgets(buf, BUFSIZE-1, stdin) == NULL) break;          \
    }                                                           \
  }

#define func_f2_f(funcStr, funcName) {                          \
    while (startsWith(buf, funcStr " ")) {                      \
      uint32_t u;                                               \
      sscanf(buf, funcStr " %x", &u);                           \
      *b0 = u2f(u);                                             \
      funcName<<<1, 1>>>(s2, b0);                               \
      hipDeviceSynchronize();                                  \
      printf("%x %x\n", f2u(s2->x), f2u(s2->y));                \
      fflush(stdout);                                           \
      if (fgets(buf, BUFSIZE-1, stdin) == NULL) break;          \
    }                                                           \
  }

#define func_f_f_f(funcStr, funcName) {                         \
    while (startsWith(buf, funcStr " ")) {                      \
      uint32_t u, v;                                            \
      sscanf(buf, funcStr " %x %x", &u, &v);                    \
      *b0 = u2f(u);                                             \
      *b1 = u2f(v);                                             \
      funcName<<<1, 1>>>(s, b0, b1);                            \
      hipDeviceSynchronize();                                  \
      printf("%x\n", f2u(*s));                                  \
      fflush(stdout);                                           \
      if (fgets(buf, BUFSIZE-1, stdin) == NULL) break;          \
    }                                                           \
  }

//

#define BUFSIZE 1024

int main(int argc, char **argv) {
#if 0
  hipInit(0);

  int ndevice;
  hipGetDeviceCount(&ndevice);
  if (ndevice == 0) {
    fprintf(stderr, "No cuda device available\n");
    exit(0);
  }

  hipDevice_t device;
  char deviceName[1024];
  hipDeviceGet(&device, 0);
  hipDeviceGetName(deviceName, 1000, device);
  fprintf(stderr, "Device : %s\n", deviceName);
#endif

  hipSetDeviceFlags(hipDeviceScheduleSpin);

  vdouble2 *r2;
  vfloat2 *s2;
  double *r, *a0, *a1, *a2;
  float *s, *b0, *b1, *b2;
  int *i0;
  hipMallocManaged(&r , 1*sizeof(double));
  hipMallocManaged(&r2, 1*sizeof(vdouble2));
  hipMallocManaged(&a0, 1*sizeof(double));
  hipMallocManaged(&a1, 1*sizeof(double));
  hipMallocManaged(&a2, 1*sizeof(double));
  hipMallocManaged(&s , 1*sizeof(float));
  hipMallocManaged(&s2, 1*sizeof(vfloat2));
  hipMallocManaged(&b0, 1*sizeof(float));
  hipMallocManaged(&b1, 1*sizeof(float));
  hipMallocManaged(&b2, 1*sizeof(float));
  hipMallocManaged(&i0, 1*sizeof(int));

  printf("3\n");
  fflush(stdout);

  char buf[BUFSIZE];
  if (fgets(buf, BUFSIZE-1, stdin)) {}

  while(!feof(stdin)) {
    func_d_d("sin", xsin);
    func_d_d("cos", xcos);
    func_d_d("tan", xtan);
    func_d_d("asin", xasin);
    func_d_d("acos", xacos);
    func_d_d("atan", xatan);
    func_d_d("log", xlog);
    func_d_d("exp", xexp);

    func_d_d("sqrt", xsqrt);
    func_d_d("sqrt_u05", xsqrt_u05);
    func_d_d("sqrt_u35", xsqrt_u35);
    func_d_d("cbrt", xcbrt);
    func_d_d("cbrt_u1", xcbrt_u1);

    func_d_d("sinh", xsinh);
    func_d_d("cosh", xcosh);
    func_d_d("tanh", xtanh);
    func_d_d("sinh_u35", xsinh_u35);
    func_d_d("cosh_u35", xcosh_u35);
    func_d_d("tanh_u35", xtanh_u35);
    func_d_d("asinh", xasinh);
    func_d_d("acosh", xacosh);
    func_d_d("atanh", xatanh);

    func_d_d("sin_u1", xsin_u1);
    func_d_d("cos_u1", xcos_u1);
    func_d_d("tan_u1", xtan_u1);
    func_d_d("sinpi_u05", xsinpi_u05);
    func_d_d("cospi_u05", xcospi_u05);
    func_d_d("asin_u1", xasin_u1);
    func_d_d("acos_u1", xacos_u1);
    func_d_d("atan_u1", xatan_u1);
    func_d_d("log_u1", xlog_u1);

    func_d_d("exp2", xexp2);
    func_d_d("exp10", xexp10);
    func_d_d("exp2_u35", xexp2_u35);
    func_d_d("exp10_u35", xexp10_u35);
    func_d_d("expm1", xexpm1);
    func_d_d("log10", xlog10);
    func_d_d("log2", xlog2);
    func_d_d("log2_u35", xlog2_u35);
    func_d_d("log1p", xlog1p);
    func_d_d("fabs", xfabs);
    func_d_d("trunc", xtrunc);
    func_d_d("floor", xfloor);
    func_d_d("ceil", xceil);
    func_d_d("round", xround);
    func_d_d("rint", xrint);
    func_d_d("frfrexp", xfrfrexp);
    func_d_d("tgamma_u1", xtgamma_u1);
    func_d_d("lgamma_u1", xlgamma_u1);
    func_d_d("erf_u1", xerf_u1);
    func_d_d("erfc_u15", xerfc_u15);

    func_d2_d("sincos", xsincos);
    func_d2_d("sincos_u1", xsincos_u1);
    func_d2_d("sincospi_u35", xsincospi_u35);
    func_d2_d("sincospi_u05", xsincospi_u05);
    func_d2_d("modf", xmodf);

    func_d_d_d("pow", xpow);
    func_d_d_d("atan2", xatan2);
    func_d_d_d("atan2_u1", xatan2_u1);
    func_d_d_d("hypot_u05", xhypot_u05);
    func_d_d_d("hypot_u35", xhypot_u35);
    func_d_d_d("copysign", xcopysign);
    func_d_d_d("fmax", xfmax);
    func_d_d_d("fmin", xfmin);
    func_d_d_d("fdim", xfdim);
    func_d_d_d("nextafter", xnextafter);
    func_d_d_d("fmod", xfmod);
    func_d_d_d("remainder", xremainder);

    func_d_d_i("ldexp", xldexp);
    func_i_d("ilogb", xilogb);
    func_i_d("expfrexp", xexpfrexp);

    //

    func_f_f("sinf", xsinf);
    func_f_f("cosf", xcosf);
    func_f_f("tanf", xtanf);
    func_f_f("asinf", xasinf);
    func_f_f("acosf", xacosf);
    func_f_f("atanf", xatanf);
    func_f_f("logf", xlogf);
    func_f_f("expf", xexpf);

    func_f_f("sqrtf", xsqrtf);
    func_f_f("sqrtf_u05", xsqrtf_u05);
    func_f_f("sqrtf_u35", xsqrtf_u35);
    func_f_f("cbrtf", xcbrtf);
    func_f_f("cbrtf_u1", xcbrtf_u1);

    func_f_f("sinhf", xsinhf);
    func_f_f("coshf", xcoshf);
    func_f_f("tanhf", xtanhf);
    func_f_f("sinhf_u35", xsinhf_u35);
    func_f_f("coshf_u35", xcoshf_u35);
    func_f_f("tanhf_u35", xtanhf_u35);
    func_f_f("asinhf", xasinhf);
    func_f_f("acoshf", xacoshf);
    func_f_f("atanhf", xatanhf);

    func_f_f("sinf_u1", xsinf_u1);
    func_f_f("cosf_u1", xcosf_u1);
    func_f_f("tanf_u1", xtanf_u1);
    func_f_f("sinpif_u05", xsinpif_u05);
    func_f_f("cospif_u05", xcospif_u05);
    func_f_f("asinf_u1", xasinf_u1);
    func_f_f("acosf_u1", xacosf_u1);
    func_f_f("atanf_u1", xatanf_u1);
    func_f_f("logf_u1", xlogf_u1);

    func_f_f("exp2f", xexp2f);
    func_f_f("exp10f", xexp10f);
    func_f_f("exp2f_u35", xexp2f_u35);
    func_f_f("exp10f_u35", xexp10f_u35);
    func_f_f("expm1f", xexpm1f);
    func_f_f("log10f", xlog10f);
    func_f_f("log2f", xlog2f);
    func_f_f("log2f_u35", xlog2f_u35);
    func_f_f("log1pf", xlog1pf);

    func_f2_f("sincosf", xsincosf);
    func_f2_f("sincosf_u1", xsincosf_u1);
    func_f2_f("sincospif_u35", xsincospif_u35);
    func_f2_f("sincospif_u05", xsincospif_u05);

    func_f_f_f("powf", xpowf);
    func_f_f_f("atan2f", xatan2f);
    func_f_f_f("atan2f_u1", xatan2f_u1);

    func_f_f("fabsf", xfabsf);
    func_f_f("truncf", xtruncf);
    func_f_f("floorf", xfloorf);
    func_f_f("ceilf", xceilf);
    func_f_f("roundf", xroundf);
    func_f_f("rintf", xrintf);
    func_f_f("frfrexpf", xfrfrexpf);

    func_f_f_f("hypotf_u05", xhypotf_u05);
    func_f_f_f("hypotf_u35", xhypotf_u35);
    func_f_f_f("copysignf", xcopysignf);
    func_f_f_f("fmaxf", xfmaxf);
    func_f_f_f("fminf", xfminf);
    func_f_f_f("fdimf", xfdimf);
    func_f_f_f("nextafterf", xnextafterf);
    func_f_f_f("fmodf", xfmodf);
    func_f_f_f("remainderf", xremainderf);

    func_f2_f("modff", xmodff);

    func_f_f("tgammaf_u1", xtgammaf_u1);
    func_f_f("lgammaf_u1", xlgammaf_u1);
    func_f_f("erff_u1", xerff_u1);
    func_f_f("erfcf_u15", xerfcf_u15);

    func_f_f("fastsinf_u3500", xfastsinf_u3500);
    func_f_f("fastcosf_u3500", xfastcosf_u3500);
    func_f_f_f("fastpowf_u3500", xfastpowf_u3500);
  }

  return 0;
}
